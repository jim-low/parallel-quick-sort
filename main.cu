#include <iostream>
#include <ctime>
#include <vector>
#include <mpi.h>
#include <omp.h>
#include "QuickSort.h"
#include "OMPParallelQuickSort.h"
#include "MPIParallelQuickSort.h"
#include "CUDAParallelQuickSort.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <chrono>

float* generate_float_array(size_t size)
{
	float* arr = (float*)calloc(size, sizeof(float));

	for (int i = 0; i < size; ++i)
	{
		arr[i] = ((float)rand()) / ((float)rand());
	}

	return arr;
}

float* generate_descending_array(size_t size) 
{
	int count;
	float* arr = (float*)calloc(size, sizeof(float));
	count = size;
	for (int i = 0; i < size; ++i)
	{
		count--;
		arr[i] = count;
	}
	return arr;
}

int main(int argc, char** argv)
{
	srand(time(0));
	size_t size = 10000000;
	float* arr = generate_float_array(size);

	//for (int i = 0; i < size; ++i)
	//{
	//	std::cout << arr[i] << " ";
	//}
	//std::cout << std::endl;
	//std::cout << std::endl;
	//std::cout << std::endl;

	// MPI Parallel Quick Sort
	//MPI_Init(&argc, &argv);
	//MPIParallelQuickSort mpiSort = MPIParallelQuickSort(arr, size);
	//mpiSort.sort();
	//MPI_Finalize();

	// OpenMP Part (need to comment the upper part to run)
	//double runtime;
	//omp_set_num_threads(8);

	//OMPParallelQuickSort ompSort = OMPParallelQuickSort(arr, size);
	//runtime = omp_get_wtime();
	//ompSort.sort();
	////ompSort.display();

	//runtime = omp_get_wtime() - runtime;
	//// std::cout << "\n\nUsed " << runtime << " seconds." << std::endl;
	//printf("\nUsed %.9f seconds.\n\n", runtime);

	//// Part for used to compare
	//OMPParallelQuickSort ompSort2 = OMPParallelQuickSort(arr, size);
	//runtime = omp_get_wtime();
	//ompSort2.sort2();
	////ompSort2.display();

	//runtime = omp_get_wtime() - runtime;

	//printf("\nUsed %.9f seconds without OpenMP.\n\n", runtime);

	// OpenMP Part (need to comment the upper part to run)
	//double runtime;
	//omp_set_num_threads(8);

	//OMPParallelQuickSort ompSort = OMPParallelQuickSort(arr, size);
	//runtime = omp_get_wtime();
	//ompSort.sort();
	////ompSort.display();

	//runtime = omp_get_wtime() - runtime;
	//// std::cout << "\n\nUsed " << runtime << " seconds." << std::endl;
	//printf("\nUsed %.9f seconds.\n\n", runtime);

	//// Part for used to compare
	//QuickSort ompSort2 = QuickSort(arr, size);
	//runtime = omp_get_wtime();
	//ompSort2.sort();
	////ompSort2.display();

	//runtime = omp_get_wtime() - runtime;

	//printf("\nUsed %.9f seconds without OpenMP.\n\n", runtime);

	//CUDA
	//size_t free_memory, total_memory;
	//hipError_t cudaStatus = hipMemGetInfo(&free_memory, &total_memory);
	//if (cudaStatus != hipSuccess) {
	//	std::cerr << "hipMemGetInfo failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
	//	// Handle the error
	//}
	//else {
	//	std::cout << "CUDA: Total GPU Memory: " << total_memory / (1024 * 1024) << " MB" << std::endl;
	//	std::cout << "CUDA: Free GPU Memory: " << free_memory / (1024 * 1024) << " MB" << std::endl;
	//}

	//size_t data_size_bytes = size * sizeof(float);
	//double data_size_mb = static_cast<double>(data_size_bytes) / (1024 * 1024);
	//std::cout << "CUDA: Requirement to sort array: " << data_size_mb << " MB" << std::endl;


	//CUDA Parallel Quick Sort
	if (arr != nullptr) {

		QuickSort standardSorter(arr, size);

		CUDAParallelQuickSort CUDAsorter(arr, size);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);
		CUDAsorter.sort();
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("CUDA Sort Duration: %.2f ms\n", milliseconds);

		//CUDAsorter.display();

		auto start_time = std::chrono::high_resolution_clock::now();
		standardSorter.sort();
		auto end_time = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> elapsed_time = end_time - start_time;
		std::cout << "\n\n Standard Sort Duration: " << elapsed_time.count() << "ms" << std::endl;
		//standardSorter.display();
		
	}

	else {
		std::cerr << "Error: Failed to allocate memory for arr." << std::endl;
	}


	free(arr);
	return 0;
}
