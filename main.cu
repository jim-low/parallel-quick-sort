#include <iostream>
#include <ctime>
#include <vector>
#include <mpi.h>
#include <omp.h>
#include "QuickSort.h"
#include "OMPParallelQuickSort.h"
#include "MPIParallelQuickSort.h"
#include "CUDAParallelQuickSort.cuh"
#include "hip/hip_runtime.h"
#include ""

float* generate_float_array(size_t size)
{
	float* arr = (float*)calloc(size, sizeof(float));

	for (int i = 0; i < size; ++i)
	{
		arr[i] = ((float)rand()) / ((float)rand());
	}

	return arr;
}

int main(int argc, char** argv)
{

	srand(time(0));
	size_t size = 10;
	float* arr = generate_float_array(size);

	//for (int i = 0; i < size; ++i)
	//{
	//	std::cout << arr[i] << " ";
	//}
	//std::cout << std::endl;
	//std::cout << std::endl;
	//std::cout << std::endl;

	// MPI Parallel Quick Sort
	//MPI_Init(&argc, &argv);
	//MPIParallelQuickSort mpiSort = MPIParallelQuickSort(arr, size);
	//mpiSort.sort();
	//MPI_Finalize();

	// OpenMP Part (need to comment the upper part to run)
	//double runtime;
	//omp_set_num_threads(8);

	//OMPParallelQuickSort ompSort = OMPParallelQuickSort(arr, size);
	//runtime = omp_get_wtime();
	//ompSort.sort();
	////ompSort.display();

	//runtime = omp_get_wtime() - runtime;
	//// std::cout << "\n\nUsed " << runtime << " seconds." << std::endl;
	//printf("\nUsed %.9f seconds.\n\n", runtime);

	//// Part for used to compare
	//OMPParallelQuickSort ompSort2 = OMPParallelQuickSort(arr, size);
	//runtime = omp_get_wtime();
	//ompSort2.sort2();
	////ompSort2.display();

	//runtime = omp_get_wtime() - runtime;

	//printf("\nUsed %.9f seconds without OpenMP.\n\n", runtime);

	//CUDA
	
	//CUDA Parallel Quick Sort
	if (arr != nullptr) {

		hipError_t cudaStatus = hipStreamQuery(0);
		if (cudaStatus != hipSuccess) {
			printf("CUDA IS NOT FREED.\n");
		}

		QuickSort standardSorter(arr, size);

		CUDAParallelQuickSort CUDAsorter(arr, size);

		hipFree(0);
		hipError_t cudaStatus1 = hipStreamQuery(0);
		if (cudaStatus1 != hipSuccess) {
			printf("Stream 0 is OCCUPIED.\n");
		}

		CUDAsorter.sort();

		CUDAsorter.display();

		standardSorter.sort();

		standardSorter.display();
		
	}
	else {
		std::cerr << "Error: Failed to allocate memory for arr." << std::endl;
	}


	free(arr);
	return 0;
}
