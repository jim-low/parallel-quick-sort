#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "CUDAParallelQuickSort.cuh"


//constructor
CUDAParallelQuickSort::CUDAParallelQuickSort(float* arr, size_t size)
{
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    hipError_t err = hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 24);
    if (err != hipSuccess) {
        std::cerr << "hipDeviceSetLimit failed! Error: " << hipGetErrorString(err) << std::endl;
        // Handle the error
        return;
    }

    this->size = size;

    this->h_unsorted = (float*)calloc(size, sizeof(float));
    this->h_sorted = (float*)calloc(size, sizeof(float));

    hipMalloc((void**)&this->d_unsorted, size * sizeof(float));
    hipMalloc((void**)&this->d_sorted, size * sizeof(float));

    hipMemcpy((void*)this->d_unsorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)this->d_sorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
}

//deconstructor
CUDAParallelQuickSort::~CUDAParallelQuickSort()
{
    free(this->h_unsorted);
    free(this->h_sorted);

    hipFree(this->d_unsorted);
    hipFree(this->d_sorted);
}


//CUDA kernel for partitioning
__device__ float partition(float* arr, int low, int high)
{
    float pivot = arr[high];

    int swapMarker = low - 1;

    for (int j = low; j < high; ++j) {
        if (arr[j] <= pivot) {
            ++swapMarker;
            float temp = arr[swapMarker];
            arr[swapMarker] = arr[j];
            arr[j] = temp;
        }
    }

    float temp = arr[swapMarker + 1];
    arr[swapMarker + 1] = arr[high];
    arr[high] = temp;

    return swapMarker + 1;
}

__device__ void quickSort(float* arr, int left, int right) {

    if (left < right) {
        int pivotIndex = partition(arr, left, right);
        quickSort(arr, left, pivotIndex - 1);
        quickSort(arr, pivotIndex + 1, right);
    }
}

//global functions can be called from the host and executed on the device.
__global__ void cudaQuicksort(float* arr, int left, int right, int maxDepth) {
    
    int pivotIndex = partition(arr, left, right); //getting the pivot index and initiating the partition process

    if (left < right)
    {
        if (maxDepth > 0) {
            //hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
            cudaQuicksort <<<1, 8>>> (arr, left, pivotIndex - 1, maxDepth - 1);

            //hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
            cudaQuicksort <<<1, 8>>> (arr, pivotIndex + 1, right, maxDepth - 1);

            quickSort(arr, left, right);
            //printf("MAXDEPTH LEFT: %d.\n", maxDepth);
        
        }

        else {
            //printf("MAXDEPTH REACHED, RESORTING TO STANDARD SORT\n");
            quickSort(arr, left, right);
        }
        
        

    }
}

__host__ void CUDAParallelQuickSort::sort()
{
    cudaQuicksort<<<1, 8>>>(this->d_sorted, 0, size - 1, 16);
    hipDeviceSynchronize();
}

//display the result
void CUDAParallelQuickSort::display()
{
    hipError_t cudaStatus1 = hipStreamSynchronize(0);
    if (cudaStatus1 != hipSuccess) {
        printf("Stream 0 is not done yet.\n");
        return;
    }

    //hipEventRecord(start);
    hipError_t cudaStatus = hipMemcpy(this->h_unsorted, this->d_unsorted, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    cudaStatus = hipMemcpy(this->h_sorted, this->d_sorted, size * sizeof(float), hipMemcpyDeviceToHost); 
    if (cudaStatus != hipSuccess){
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
    }


    printf("Unsorted Array:\n");
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_unsorted[i] << " ";
    }
    std::cout << std::endl;
    std::cout << std::endl;

    std::cout << "Sorted Array:" << std::endl;
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_sorted[i] << " ";
    }
    std::cout << std::endl;
}



