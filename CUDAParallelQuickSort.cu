#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "CUDAParallelQuickSort.cuh"


//constructor
CUDAParallelQuickSort::CUDAParallelQuickSort(float* arr, size_t size)
{
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    this->size = size;

    this->h_unsorted = (float*)calloc(size, sizeof(float));
    this->h_sorted = (float*)calloc(size, sizeof(float));

    hipMalloc((void**)&this->d_unsorted, size * sizeof(float));
    hipMalloc((void**)&this->d_sorted, size * sizeof(float));

    hipMemcpy((void*)this->d_unsorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)this->d_sorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
}

//deconstructor
CUDAParallelQuickSort::~CUDAParallelQuickSort()
{
    hipFree(this->d_unsorted);
    hipFree(this->d_sorted);
    hipStreamDestroy(0);

    free(this->h_unsorted);
    free(this->h_sorted);
}


//CUDA kernel for partitioning
__device__ float partition(float* arr, int low, int high)
{
    float pivot = arr[high];

    int swapMarker = low - 1;

    for (int j = low; j < high; ++j) {
        if (arr[j] <= pivot) {
            ++swapMarker;
            float temp = arr[swapMarker];
            arr[swapMarker] = arr[j];
            arr[j] = temp;
        }
    }

    float temp = arr[swapMarker + 1];
    arr[swapMarker + 1] = arr[high];
    arr[high] = temp;

    return swapMarker + 1;
}

//global functions can be called from the host and executed on the device.
__global__ void cudaQuicksort(float* arr, int left, int right) {
    
    int pivotIndex = partition(arr, left, right); //getting the pivot index and initiating the partition process

    if (left < pivotIndex - 1)
    {
        //hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cudaQuicksort <<<1, 8>>> (arr, left, pivotIndex - 1);

    }
   
    if (right > pivotIndex + 1)
    {
        //hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        cudaQuicksort <<<1, 8>>> (arr, pivotIndex + 1, right);

    }

    //if (left < right)
    //{
    //    int pivotIndex = partition(arr, left, right); //getting the pivot index and initiating the partition process

    //    cudaQuicksort <<<1, 8 >>> (arr, left, pivotIndex - 1);
    //    cudaQuicksort <<<1, 8 >>> (arr, pivotIndex + 1, right);
    //}


   //__syncthreads();
}

//__host__ void callKernel(hipStream_t stream) {
//    cudaQuicksort << <1, 8, 0, stream >> > (arr, pivotIndex + 1, right);
//}

//preparing stream 
__host__ void CUDAParallelQuickSort::sort()
{
    //hipStream_t s1, s2; //initializing stream

    ////preset cudaStream, initilaize s1 to the new Stream, Set flag to be able to overlap other streams
    //hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    //hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);

    //hipStreamCreate(&s1);
    //hipStreamCreate(&s2);

    cudaQuicksort<<<1, 8>>>(this->d_sorted, 0, size - 1);

    hipStreamDestroy(0);
    

    //hipStreamSynchronize(s1);
    //hipStreamSynchronize(s2);

    //hipStreamDestroy(s1);
}

//display the result
void CUDAParallelQuickSort::display()
{
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    hipMemcpy(this->h_unsorted, this->d_unsorted, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->h_sorted, this->d_sorted, size * sizeof(float), hipMemcpyDeviceToHost);

    printf("Unsorted Array:\n");
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_unsorted[i] << " ";
    }
    std::cout << std::endl;
    std::cout << std::endl;

    std::cout << "Sorted Array:" << std::endl;
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_sorted[i] << " ";
    }
    std::cout << std::endl;
}



