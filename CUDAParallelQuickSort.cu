#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "CUDAParallelQuickSort.cuh"


//constructor
CUDAParallelQuickSort::CUDAParallelQuickSort(float* arr, size_t size)
{
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    this->size = size;

    this->h_unsorted = (float*)calloc(size, sizeof(float));
    this->h_sorted = (float*)calloc(size, sizeof(float));

    hipMalloc((void**)&this->d_unsorted, size * sizeof(float));
    hipMalloc((void**)&this->d_sorted, size * sizeof(float));

    hipMemcpy((void*)this->d_unsorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)this->d_sorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
}

//deconstructor
CUDAParallelQuickSort::~CUDAParallelQuickSort()
{
    free(this->h_unsorted);
    free(this->h_sorted);

    hipFree(this->d_unsorted);
    hipFree(this->d_sorted);
}


//CUDA kernel for partitioning
__device__ float partition(float* arr, int low, int high)
{
    float pivot = arr[high];

    int swapMarker = low - 1;

    for (int j = low; j < high; ++j) {
        if (arr[j] <= pivot) {
            ++swapMarker;
            float temp = arr[swapMarker];
            arr[swapMarker] = arr[j];
            arr[j] = temp;
        }
    }

    float temp = arr[swapMarker + 1];
    arr[swapMarker + 1] = arr[high];
    arr[high] = temp;

    return swapMarker + 1;
}

__device__ void quickSort(float* arr, int left, int right) {

    if (left < right) {
        int pivotIndex = partition(arr, left, right);
        quickSort(arr, left, pivotIndex - 1);
        quickSort(arr, pivotIndex + 1, right);
    }
}


//global functions can be called from the host and executed on the device.
__global__ void cudaQuicksort(float* arr, int left, int right) {
    __syncthreads();

    int pivotIndex = partition(arr, left, right); //getting the pivot index and initiating the partition process

    hipStream_t anotherFuckingStream;
    hipStreamCreateWithFlags(&anotherFuckingStream, hipStreamNonBlocking);

    if (left < pivotIndex - 1)
    {
        cudaQuicksort << <1, 8, 0, anotherFuckingStream >> > (arr, left, pivotIndex - 1);
    }

    if (right > pivotIndex + 1)
    {
        cudaQuicksort << <1, 8, 0, anotherFuckingStream >> > (arr, pivotIndex + 1, right);
    }

    hipStreamDestroy(anotherFuckingStream);
}

__host__ void CUDAParallelQuickSort::sort()
{
    hipStream_t mainFuckingStream; //initializing stream

    //preset cudaStream, initilaize s1 to the new Stream, Set flag to be able to overlap other streams
    hipStreamCreateWithFlags(&mainFuckingStream, hipStreamNonBlocking);

    cudaQuicksort << <1, 8, 0, mainFuckingStream >> > (this->d_sorted, 0, size - 1);

    hipStreamSynchronize(mainFuckingStream);

    hipStreamDestroy(mainFuckingStream);
}

//display the result
void CUDAParallelQuickSort::display()
{
    hipError_t cudaStatus1 = hipStreamSynchronize(0);
    if (cudaStatus1 != hipSuccess) {
        printf("Stream 0 is not done yet.\n");
        return;
    }

    //hipEventRecord(start);
    hipError_t cudaStatus = hipMemcpy(this->h_unsorted, this->d_unsorted, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    cudaStatus = hipMemcpy(this->h_sorted, this->d_sorted, size * sizeof(float), hipMemcpyDeviceToHost); 
    if (cudaStatus != hipSuccess){
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
    }


    printf("Unsorted Array:\n");
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_unsorted[i] << " ";
    }
    std::cout << std::endl;
    std::cout << std::endl;

    std::cout << "Sorted Array:" << std::endl;
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_sorted[i] << " ";
    }
    std::cout << std::endl;
}



