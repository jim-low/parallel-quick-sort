#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "CUDAParallelQuickSort.cuh"

//constructor
CUDAParallelQuickSort::CUDAParallelQuickSort(float* arr, size_t size)
{
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    this->size = size;

    this->h_unsorted = (float*)calloc(size, sizeof(float));
    this->h_sorted = (float*)calloc(size, sizeof(float));

    hipMalloc((void**)&this->d_unsorted, size * sizeof(float));
    hipMalloc((void**)&this->d_sorted, size * sizeof(float));

    hipMemcpy((void*)this->d_unsorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)this->d_sorted, (void*)arr, size * sizeof(float), hipMemcpyHostToDevice);
}

//deconstructor
CUDAParallelQuickSort::~CUDAParallelQuickSort()
{
    free(this->h_unsorted);
    free(this->h_sorted);

    hipFree(this->d_unsorted);
    hipFree(this->d_sorted);

    hipDeviceReset();
}


//CUDA kernel for partitioning
__device__ float partition(float* arr, int low, int high)
{
    float pivot = arr[high];

    int swapMarker = low - 1;

    for (int j = low; j < high; ++j) {
        if (arr[j] <= pivot) {
            ++swapMarker;
            float temp = arr[swapMarker];
            arr[swapMarker] = arr[j];
            arr[j] = temp;
        }
    }

    float temp = arr[swapMarker + 1];
    arr[swapMarker + 1] = arr[high];
    arr[high] = temp;

    return swapMarker + 1;
}

__device__ void quickSort(float* arr, int left, int right) {

    if (left < right) {
        int pivotIndex = partition(arr, left, right);
        quickSort(arr, left, pivotIndex - 1);
        quickSort(arr, pivotIndex + 1, right);
    }
}


//global functions can be called from the host and executed on the device.
__global__ void cudaQuicksort(float* arr, int left, int right, int maxDepth) {

    //if array has reached a certain point, convert to standard quick sort
    if (maxDepth >= 16 || right - left <= 40) {
        quickSort(arr, left, right);
        return;
    } 

    int pivotIndex = partition(arr, left, right); //getting the pivot index and initiating the partition process


    if (left < pivotIndex - 1)
    {
        hipStream_t mainFuckingStream;
        hipStreamCreateWithFlags(&mainFuckingStream, hipStreamNonBlocking);
        cudaQuicksort <<<1, 1, 0, mainFuckingStream>>> (arr, left, pivotIndex - 1, maxDepth + 1);
        hipStreamDestroy(mainFuckingStream);
    }

    if (right > pivotIndex + 1)
    {
        hipStream_t anotherFuckingStream;
        hipStreamCreateWithFlags(&anotherFuckingStream, hipStreamNonBlocking);
        cudaQuicksort << <1, 1, 0, anotherFuckingStream >> > (arr, pivotIndex + 1, right, maxDepth + 1);
        hipStreamDestroy(anotherFuckingStream);
    }

    
    
}

__host__ void CUDAParallelQuickSort::sort()
{
    hipError_t cudaStatus = hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 16);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Failed to set Device Depth Limit! Error: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    cudaQuicksort <<<1, 1>> > (this->d_sorted, 0, size - 1, 0);
    hipDeviceSynchronize();
}

//display the result
void CUDAParallelQuickSort::display()
{

    //hipEventRecord(start);
    hipError_t cudaStatus = hipMemcpy(this->h_unsorted, this->d_unsorted, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    cudaStatus = hipMemcpy(this->h_sorted, this->d_sorted, size * sizeof(float), hipMemcpyDeviceToHost); 
    if (cudaStatus != hipSuccess){
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;
    }


    printf("Unsorted Array:\n");
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_unsorted[i] << " ";
    }
    std::cout << std::endl;
    std::cout << std::endl;

    std::cout << "Sorted Array:" << std::endl;
    for (int i = 0; i < this->size; ++i) {
        std::cout << this->h_sorted[i] << " ";
    }
    std::cout << std::endl;
}



